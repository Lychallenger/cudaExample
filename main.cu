#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;




__device__ int reduce(thread_group&g,int*x,int val){
    int lane=g.thread_rank();
//    printf("thread_rank=%d\n",lane);

    for(int i=g.size()/2;i>0;i=i/2){
        x[lane]=val;g.sync();
        val=val+x[lane+i];g.sync();
        if(lane==31){
            printf("val=%d\n",val);
        }
    }
//    printf("lane=%d,val=%d\n",lane,val);
    return val;
}
__global__ void parallel_kenel(int *x){
    int thid=threadIdx.x+blockIdx.x*blockDim.x;
    thread_block my_block = this_thread_block();//this block
//    printf("thread_rank=%d\n",my_block.thread_rank());
    auto my_tile=tiled_partition<32>(my_block);//partition 32 threads in one block;
    int result=reduce(my_tile,x+32*blockIdx.x,1);
//    printf("result=%d\n",result);

}
/**
 *
 * @param key insert key
 * @param bloom bloom filter
 * @param table hash table
 */
__global__ void insert_bloom(int*key,int*bloom,int*table){
    int thid=threadIdx.x+blockIdx.x*blockDim.x;
    int insert_key=key[thid];
    unsigned hash=(insert_key>>25)<<3;//
    //取余数
    unsigned value=0;
    for(int i=0;i<8;i++){//使用8个hash函数
        int sector_id=hash+i;
        value=(insert_key*table[i])&0x1f;//32bit
        atomicOr(bloom+sector_id,(1<<value));
    }
}
template<unsigned size>
__device__ bool search_bloom(thread_block_tile<size> g,int*key,int*bloom,int *table){
    int thid=threadIdx.x+blockIdx.x*blockDim.x;
    int lane=g.thread_rank();
    unsigned keys=key[thid/8];
    unsigned hash=(keys>>25)<<3;
    unsigned value=(keys*table[lane])&&0x1f;
    //query
    unsigned query_value=bloom[hash+lane]&(1<<value);
    unsigned tmp=g.ballot(query_value==0);
    return tmp==0;
}

__global__ void parallel_search(int*key,int*bloom,int*table){
    int thid=threadIdx.x+blockIdx.x*blockDim.x;
    thread_block my_block = this_thread_block();//this block
    auto my_tile=tiled_partition<32>(my_block);//partition 32 threads in one block;
    bool result=search_bloom(my_tile,key,bloom,table);
    if(thid%8==0){
        if(result==1)
            printf("{%d,%d}",key[thid/8],result);
    }
}
int main() {
    //test cooperative code
//    int *h_x,*d_x;
//    h_x=(int*)malloc(1024*sizeof(int));
//    memset(h_x,1,1024*sizeof(int));
//    hipMalloc(&d_x,1024*sizeof(int));
//    hipMemcpy(d_x,h_x,1024*sizeof(int),hipMemcpyHostToDevice);
//    parallel_kenel<<<3,32>>>(d_x);
//    hipDeviceSynchronize();
//    return 0;
    //test code end
    srand((unsigned int)time(0));//随机种子
    int insert_size=(1<<20);
    int*h_key,*d_key;
    h_key=(int*)malloc(insert_size*sizeof(int));
    hipMalloc(&d_key,insert_size*sizeof(int));
    for(int i=0;i<insert_size;i++){
        h_key[i]=rand()%(1<<31);
    }
    hipMemcpy(d_key,h_key,insert_size*sizeof(int),hipMemcpyHostToDevice);

    //lookup key
    int query_size=(1<<15);
    int*h_query,*d_query;
    h_query=(int*)malloc(query_size*sizeof(int));
    hipMalloc(&d_query,query_size*sizeof(int));
    for(int i=0;i<query_size;i++){
        h_query[i]=rand()%(1<<31);
    }
    hipMemcpy(d_query,h_query,query_size*sizeof(int),hipMemcpyHostToDevice);

    //bloom filter,int*;
    int bloom_size=(1<<10);//sector length
    //block length=sector length/32;
    int*h_bloom,*d_bloom;
    h_bloom=(int*)malloc(bloom_size*sizeof(int));
    hipMalloc(&d_bloom,bloom_size*sizeof(int));
    memset(h_bloom,0,bloom_size*sizeof(int));
    hipMemcpy(d_bloom,h_bloom,bloom_size*sizeof(int),hipMemcpyHostToDevice);

    //create hash table
    int*d_table;
    hipMalloc(&d_table,32*sizeof (int));
    int h_table[32]={3 ,5, 7 ,11 ,13 ,17 ,19 ,23 ,29 ,31, 37, 41, 43 ,47, 53,59,
                     61,67,71 ,73,79 ,83 ,89, 97,101,103,107,109,113,127,131,137};
    hipMemcpy(d_table,h_table,32*sizeof(int),hipMemcpyHostToDevice);

    insert_bloom<<<1024,1024>>>(d_key,d_bloom,d_table);
    hipDeviceSynchronize();
    parallel_search<<<1024,256>>>(d_query,d_bloom,d_table);
    hipDeviceSynchronize();
    return 0;
}
